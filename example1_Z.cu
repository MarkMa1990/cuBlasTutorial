#include "hip/hip_runtime.h"
#include "./common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <time.h>
#include <stdlib.h>

#include <iostream>
#include <iomanip>

// print Matrix content in row-major
void printArr(hipDoubleComplex *arr_temp,int M, int N);


__global__ void getLU(hipDoubleComplex *array, hipDoubleComplex *const Aarray, int M, bool is_L)
{
    unsigned int i =  blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j =  blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int ix = i * M + j;
    if ( ix < M * M)
    {
        if(is_L)
        {
            if(j>i)
                array[ix] = Aarray[ix];
            else
                array[ix].x = 0;
                array[ix].y = 0;
        }
        else
        {
            if(j<i)
                array[ix] = Aarray[ix];
            else
                array[ix].x = 0;
                array[ix].y = 0;

        }
    }
}

// get matrix L or U (signal is_L)
void getLU_host(hipDoubleComplex *array, hipDoubleComplex *data, int M, bool is_L);
// write matrix contents into a txt file, row-major
void writeMat(FILE *fp, hipDoubleComplex *array, int M);
// generate the permutation matrix
void getP_host(int *pivot, hipDoubleComplex *eye, int M);

int main(int argc, char **argv)
{

    // initialize the GPU card
    int dev = 0;
    CHECK(hipSetDevice(dev));
    dim3 block;
    dim3 grid;

    // create cublas handle
    hipblasHandle_t handle_cublas = 0;
    CHECK_CUBLAS(hipblasCreate(&handle_cublas));

    // define Matrix size
    int M, N;

    M = 5;
    N = M;

    // define GPU accelerating parameters (threads, blocks, and grids)
    block.x = 32;
    block.y = 32;
    grid.x = (M+block.x-1)/block.x;
    grid.y = (N+block.y-1)/block.y;

    hipDoubleComplex **arr_A;
    hipDoubleComplex **arr_B;

    // init 
    arr_A = new hipDoubleComplex*[1];
    arr_A[0] = new hipDoubleComplex[M*N];

    arr_B = new hipDoubleComplex*[1];
    arr_B[0] = new hipDoubleComplex[M*N];


    // create a matrix

    srand(time(NULL));

    for (int j0=0;j0<M;j0++)
        for (int i0=0;i0<N;i0++)
        {
            int temp = rand();
            int temp2 = rand();
            int ix = i0*M + j0;
            arr_A[0][ix].x = (double) temp / (double) RAND_MAX;
            arr_A[0][ix].y = (double) temp2 / (double) RAND_MAX;
        }

    //
    printf("Matrix A\n");
    printArr(arr_A[0], M, N);
    printf("\n");

    FILE *fp0 = fopen("mat_A.txt","w");
    writeMat(fp0,arr_A[0],M);
    fclose(fp0);

    // for GPU
    hipDoubleComplex **dev_arr_A = new hipDoubleComplex*[1];
    hipDoubleComplex **dev_arr_B;
    int *dev_arr_pivot;
    int *dev_inforArray;

    CHECK(hipMalloc((void **)&dev_arr_A[0], sizeof(hipDoubleComplex) * M*N));
    CHECK(hipMemcpy(dev_arr_A[0], arr_A[0], sizeof(hipDoubleComplex) * M*N, hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    // to store the ** pointer in GPU
    CHECK(hipMalloc((void **)&dev_arr_B, sizeof(hipDoubleComplex*) * 1));
    hipDeviceSynchronize();

    CHECK(hipMalloc((void **)&dev_arr_pivot,  sizeof(int) * M * 1));
    CHECK(hipMalloc((void **)&dev_inforArray, sizeof(int) * 1));

    // copy the *pointer
    CHECK(hipMemcpy(dev_arr_B, dev_arr_A, sizeof(hipDoubleComplex*) * 1, hipMemcpyHostToDevice));

    CHECK_CUBLAS(hipblasZgetrfBatched(handle_cublas, 
                M, 
                dev_arr_B,
                M,
                dev_arr_pivot,
                dev_inforArray,
                1)
            );
    
    hipDeviceSynchronize();



    CHECK(hipMemcpy(arr_B[0], dev_arr_A[0], sizeof(hipDoubleComplex) * M * N, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();


    //
    printf("Matrix Aarray\n");
    printArr(arr_B[0], M, N);
    printf("\n");

    // get LU
    hipDoubleComplex *mat_L = new hipDoubleComplex[M*N];
    hipDoubleComplex *mat_U = new hipDoubleComplex[M*N];
    getLU_host(mat_L, arr_B[0], M, true);
    getLU_host(mat_U, arr_B[0], M, false);


    FILE *fp1 = fopen("mat_L.txt","w");
    FILE *fp2 = fopen("mat_U.txt","w");

    writeMat(fp1,mat_L,M);
    writeMat(fp2,mat_U,M);

    fclose(fp1);
    fclose(fp2);


    int *infoHost = new int[1];
    CHECK(hipMemcpy(infoHost,dev_inforArray,sizeof(int)*1,hipMemcpyDeviceToHost));

    std::cout << "info: " << infoHost[0] << std::endl;

    // pivoting
    int *pivotHost = new int [M];
    CHECK(hipMemcpy(pivotHost, dev_arr_pivot, sizeof(int)*M,hipMemcpyDeviceToHost));

    std::cout << "the pivoting array" << std::endl;
    for (int i=0;i<M;i++)
    {
        std::cout<<pivotHost[i]<<std::endl;
    }

    hipDoubleComplex *pivot_matrix = new hipDoubleComplex[M*M];

    getP_host(pivotHost,pivot_matrix, M);
    fp1 = fopen("mat_P.txt","w");
    writeMat(fp1,pivot_matrix,M);
    fclose(fp1);

    fp1 = fopen("pivot.txt","w");
    for (int i=0;i<M;i++)
    {
        fprintf(fp1,"%d\n",pivotHost[i]);
    }
    fclose(fp1);



    // finish
    CHECK_CUBLAS(hipblasDestroy(handle_cublas));



}





void printArr(hipDoubleComplex *arr_temp,int M, int N)
{

    for (int i0=0;i0<N;i0++)
    {
        for (int j0=0;j0<M;j0++)
        {
            int ix = j0*M + i0;

            if(j0==0)
                std::cout <<"| ";
            else
                NULL;

            std::cout << std::fixed <<  std::setprecision(6) << std::setfill('0') << arr_temp[ix].x << "+j" << arr_temp[ix].y << ", ";

            if(j0==M-1)
                std::cout <<"|";
            else
                NULL;
        }
        std::cout << std::endl;

    }
    std::cout << std::endl;


}


void getLU_host(hipDoubleComplex *array, hipDoubleComplex *data, int M, bool is_L)
{
    for(int j=0;j<M;j++)
        for(int i=0;i<M;i++)
        {
            if(is_L)
            {
                int ix = j*M+i;
                if (i>j)
                    array[ix] = data[ix];
                else if (i==j)
                {
                    array[ix].x = 1;
                    array[ix].y = 0;
                }
                else
                {
                    array[ix].x = 0;
                    array[ix].y = 0;
                }
            }
            else
            {
                int ix = j*M+i;
                if (i<=j)
                    array[ix] = data[ix];
                else
                {
                    array[ix].x = 0;
                    array[ix].y = 0;
                }
            }
            

        }

}


void writeMat(FILE *fp, hipDoubleComplex *array, int M)
{
    for(int i=0;i<M;i++)
        for(int j=0;j<M;j++)
        {
            int ix = j*M+i;

            fprintf(fp,"%.6f+%0.6fj",array[ix].x,array[ix].y);
            if(j==M-1)
                fprintf(fp,"\n");
            else
                fprintf(fp,",");

        }
}


void getP_host(int *pivot, hipDoubleComplex *eye, int M)
{
    // make an eye matrix
    for(int j=0;j<M;j++)
        for (int i=0;i<M;i++)
        {
            int ix = j*M+i;
            if (i==j)
            {
                eye[ix].x = 1;
                eye[ix].y = 0;
            }
            else
            {
                eye[ix].x = 0;
                eye[ix].y = 0;
            }
        }

    for (int i=0;i<M;i++)
    {
        // swap rows
        for (int j=0;j<M;j++) 
        {
            int ix = j*M+i;
            int ix_pivot = pivot[i] - 1 + j*M;
            hipDoubleComplex temp = eye[ix];
            eye[ix] = eye[ix_pivot];
            eye[ix_pivot] = temp;
        }
    }
}
